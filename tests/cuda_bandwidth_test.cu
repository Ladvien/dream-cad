#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MEMCOPY_ITERATIONS 100
#define DEFAULT_SIZE (32 * (1 << 20)) // 32 MB

void checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s - %s\n", message, hipGetErrorString(error));
        exit(1);
    }
}

float testDeviceToDeviceBandwidth(int memSize) {
    float elapsedTimeInMs = 0.0f;
    float bandwidthInGBs = 0.0f;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate device memory
    unsigned char *d_src, *d_dst;
    checkCudaError(hipMalloc((void**)&d_src, memSize), "Failed to allocate source memory");
    checkCudaError(hipMalloc((void**)&d_dst, memSize), "Failed to allocate destination memory");

    // Initialize memory
    checkCudaError(hipMemset(d_src, 0, memSize), "Failed to set source memory");
    checkCudaError(hipMemset(d_dst, 0, memSize), "Failed to set destination memory");

    // Warm up
    checkCudaError(hipMemcpy(d_dst, d_src, memSize, hipMemcpyDeviceToDevice), "Warmup memcpy failed");

    // Test
    hipEventRecord(start, 0);
    for (int i = 0; i < MEMCOPY_ITERATIONS; i++) {
        checkCudaError(hipMemcpy(d_dst, d_src, memSize, hipMemcpyDeviceToDevice), "Memcpy failed");
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTimeInMs, start, stop);

    // Calculate bandwidth in GB/s
    bandwidthInGBs = ((float)(1 << 10) * memSize * MEMCOPY_ITERATIONS) / (elapsedTimeInMs * (float)(1 << 30));

    // Clean up
    hipFree(d_src);
    hipFree(d_dst);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return bandwidthInGBs;
}

float testHostToDeviceBandwidth(int memSize, bool pinned) {
    float elapsedTimeInMs = 0.0f;
    float bandwidthInGBs = 0.0f;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate host and device memory
    unsigned char *h_src, *d_dst;
    if (pinned) {
        checkCudaError(hipHostMalloc((void**)&h_src, memSize, hipHostMallocDefault), "Failed to allocate pinned host memory");
    } else {
        h_src = (unsigned char*)malloc(memSize);
        if (!h_src) {
            fprintf(stderr, "Failed to allocate host memory\n");
            exit(1);
        }
    }
    checkCudaError(hipMalloc((void**)&d_dst, memSize), "Failed to allocate device memory");

    // Initialize memory
    memset(h_src, 0, memSize);

    // Warm up
    checkCudaError(hipMemcpy(d_dst, h_src, memSize, hipMemcpyHostToDevice), "Warmup memcpy failed");

    // Test
    hipEventRecord(start, 0);
    for (int i = 0; i < MEMCOPY_ITERATIONS; i++) {
        checkCudaError(hipMemcpy(d_dst, h_src, memSize, hipMemcpyHostToDevice), "Memcpy failed");
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTimeInMs, start, stop);

    // Calculate bandwidth in GB/s
    bandwidthInGBs = ((float)(1 << 10) * memSize * MEMCOPY_ITERATIONS) / (elapsedTimeInMs * (float)(1 << 30));

    // Clean up
    if (pinned) {
        hipHostFree(h_src);
    } else {
        free(h_src);
    }
    hipFree(d_dst);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return bandwidthInGBs;
}

float testDeviceToHostBandwidth(int memSize, bool pinned) {
    float elapsedTimeInMs = 0.0f;
    float bandwidthInGBs = 0.0f;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate host and device memory
    unsigned char *h_dst, *d_src;
    if (pinned) {
        checkCudaError(hipHostMalloc((void**)&h_dst, memSize, hipHostMallocDefault), "Failed to allocate pinned host memory");
    } else {
        h_dst = (unsigned char*)malloc(memSize);
        if (!h_dst) {
            fprintf(stderr, "Failed to allocate host memory\n");
            exit(1);
        }
    }
    checkCudaError(hipMalloc((void**)&d_src, memSize), "Failed to allocate device memory");

    // Initialize memory
    checkCudaError(hipMemset(d_src, 0, memSize), "Failed to set device memory");

    // Warm up
    checkCudaError(hipMemcpy(h_dst, d_src, memSize, hipMemcpyDeviceToHost), "Warmup memcpy failed");

    // Test
    hipEventRecord(start, 0);
    for (int i = 0; i < MEMCOPY_ITERATIONS; i++) {
        checkCudaError(hipMemcpy(h_dst, d_src, memSize, hipMemcpyDeviceToHost), "Memcpy failed");
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTimeInMs, start, stop);

    // Calculate bandwidth in GB/s
    bandwidthInGBs = ((float)(1 << 10) * memSize * MEMCOPY_ITERATIONS) / (elapsedTimeInMs * (float)(1 << 30));

    // Clean up
    if (pinned) {
        hipHostFree(h_dst);
    } else {
        free(h_dst);
    }
    hipFree(d_src);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return bandwidthInGBs;
}

int main() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        printf("No CUDA devices found!\n");
        return 1;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    
    printf("Device: %s\n", deviceProp.name);
    printf("Running bandwidth tests...\n\n");

    int memSize = DEFAULT_SIZE;
    
    // Device to Device
    float d2dBandwidth = testDeviceToDeviceBandwidth(memSize);
    printf("Device to Device Bandwidth: %.2f GB/s\n", d2dBandwidth);
    
    // Host to Device (Pageable)
    float h2dPageableBandwidth = testHostToDeviceBandwidth(memSize, false);
    printf("Host to Device Bandwidth (Pageable): %.2f GB/s\n", h2dPageableBandwidth);
    
    // Host to Device (Pinned)
    float h2dPinnedBandwidth = testHostToDeviceBandwidth(memSize, true);
    printf("Host to Device Bandwidth (Pinned): %.2f GB/s\n", h2dPinnedBandwidth);
    
    // Device to Host (Pageable)
    float d2hPageableBandwidth = testDeviceToHostBandwidth(memSize, false);
    printf("Device to Host Bandwidth (Pageable): %.2f GB/s\n", d2hPageableBandwidth);
    
    // Device to Host (Pinned)
    float d2hPinnedBandwidth = testDeviceToHostBandwidth(memSize, true);
    printf("Device to Host Bandwidth (Pinned): %.2f GB/s\n", d2hPinnedBandwidth);
    
    printf("\n");
    
    // Check if we meet RTX 3090 expected bandwidth (around 936 GB/s)
    if (d2dBandwidth > 900.0f) {
        printf("Test Result: PASS - Memory bandwidth meets RTX 3090 specifications\n");
        return 0;
    } else {
        printf("Test Result: WARNING - Memory bandwidth (%.2f GB/s) is lower than expected for RTX 3090\n", d2dBandwidth);
        return 0; // Still return 0 as it's not a failure, just a warning
    }
}